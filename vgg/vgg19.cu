#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "util.h"

#include "cudnnFunction.h"
#include "cublasFunction.h"
#include "cudaFunction.h"
#include "persistFunction.h"
#include "persistInfer.h"

__global__ void isSame(float *array1, float *array2, int height, int width, bool transform)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadId >= height * width)
	{
		return;
	}

	int r = threadId / width;
	int c = threadId % width;

	//if(threadId > 1024) // for debugging
	//{
	//	return;
	//}

	if(transform)
	{
		if(fabsf(array1[r * width + c] - array2[c * height + r]) > 1e-3)
		{
			printf("not same at %d, %f ~ %f\n", 
					threadId, array1[r * width + c], array2[c * height + r]);
		}
	}
	else
	{
		if(fabsf(array1[r * width + c] - array2[r * width + c]) > 1e-3)
		{
			printf("not same at %d, %f ~ %f\n", 
					threadId, array1[r * width + c], array2[r * width + c]);
		}
		//else
		//{
		//	printf("%d: %f ~ %f\n", 
		//			threadId, array1[r * width + c], array2[r * width + c]);
		//}
	}
}

__global__ void printArray(float *array, int nBatch, int nChannels, int height, int width, int n)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadId >= n)
	{
		return;
	}

	int posN = threadId / (nChannels * height * width);
	int posC = threadId % (nChannels * height * width) / (height * width);
	int posH = threadId % (height * width) / width;
	int posW = threadId % (height * width) % width;

	printf("%d-%d-%d-%d: %f\n", posN, posC, posH, posW, array[threadId]);
}

int main(int argc, char **argv)
{
	printf("num args: %d\n", argc);
	char *image_file = argv[1];
	char *weights_file = argv[2];
	char *bias_file = argv[3];
	char *output_file = argv[4];

	struct timeval start, end;

	int blockDim = 256;
	int gridDim;

	CNNFunction *func = new CNNPersistFunction();
	func->init();
	func->readImage(image_file);
	func->readParameters(weights_file, bias_file);

	//printArray<<<gridDim, blockDim>>>(func->featureOut, 1, theFilters, theWidth, theWidth, theFilters * theWidth * theWidth);
	//checkCudaErrors(hipDeviceSynchronize());
	//func->convolution(224, 3, 64, 0);
	//funcCudnn->convolution(224, 3, 64, 0);

	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&start, NULL);

	func->convPersist(224, 3, 64, 0);
	func->convPersist(224, 64, 64, 1);
	
	PersistInfer::signalIn[0] = 1;
	__sync_synchronize();

#ifdef DEBUG
	checkCudaErrors(hipDeviceSynchronize());

	CNNFunction *funcCudnn = new CNNCudnnFunction();
	funcCudnn->init();
	funcCudnn->readImage(image_file);
	funcCudnn->readParameters(weights_file, bias_file);

	funcCudnn->convolution(224, 3, 64, 0);
	funcCudnn->convolution(224, 64, 64, 1);

	int theWidth = 224;
	int theFilters = 64;
	checkCudaErrors(hipDeviceSynchronize());

	gridDim = (theWidth * theWidth * theFilters + blockDim - 1) / blockDim;
	isSame<<<gridDim, blockDim>>>(func->featureMap[2], funcCudnn->featureOut, theWidth * theWidth, theFilters, false);
	checkCudaErrors(hipDeviceSynchronize());

	return 0;
#endif

	func->maxpool(224, 64);
	//funcCudnn->maxpool(224, 64);

	func->convolution(112, 64, 128, 2);
	//funcCudnn->convolution(112, 64, 128, 2);

	func->convolution(112, 128, 128, 3);
	//funcCudnn->convolution(112, 128, 128, 3);

	func->maxpool(112, 128);
	//funcCudnn->maxpool(112, 128);

	func->convolution(56, 128, 256, 4);
	//funcCudnn->convolution(56, 128, 256, 4);

	func->convolution(56, 256, 256, 5);
	//funcCudnn->convolution(56, 256, 256, 5);

	func->convolution(56, 256, 256, 6);
	//funcCudnn->convolution(56, 256, 256, 6);

	func->convolution(56, 256, 256, 7);
	//funcCudnn->convolution(56, 256, 256, 7);

	func->maxpool(56, 256);
	//funcCudnn->maxpool(56, 256);

	func->convolution(28, 256, 512, 8);
	//funcCudnn->convolution(28, 256, 512, 8);

	func->convolution(28, 512, 512, 9);
	//funcCudnn->convolution(28, 512, 512, 9);

	func->convolution(28, 512, 512, 10);
	//funcCudnn->convolution(28, 512, 512, 10);

	func->convolution(28, 512, 512, 11);
	//funcCudnn->convolution(28, 512, 512, 11);

	func->maxpool(28, 512);
	//funcCudnn->maxpool(28, 512);

	func->convolution(14, 512, 512, 12);
	//funcCudnn->convolution(14, 512, 512, 12);

	func->convolution(14, 512, 512, 13);
	//funcCudnn->convolution(14, 512, 512, 13);

	func->convolution(14, 512, 512, 14);
	//funcCudnn->convolution(14, 512, 512, 14);

	func->convolution(14, 512, 512, 15 );
	//funcCudnn->convolution(14, 512, 512, 15 );

	func->maxpool(14, 512);
	//funcCudnn->maxpool(14, 512);
	
	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&end, NULL);
	struct timeval delta = timeDelta(start, end);
	printTime(delta);

	func->fullyConnected(7, 512, 4096, 16); // most time consuming file input
	//funcCudnn->fullyConnected(7, 512, 4096, 16); // most time consuming file input

	func->fullyConnected(1, 4096, 4096, 17);
	//funcCudnn->fullyConnected(1, 4096, 4096, 17);

	func->fullyConnected(1, 4096, 1000, 18);
	//funcCudnn->fullyConnected(1, 4096, 1000, 18);

	// write 1000 dimension
	func->writeOutput(output_file);

	return 0;
}


