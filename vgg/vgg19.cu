#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cudnnFunction.h"
#include "cublasFunction.h"
#include "cudaFunction.h"

__global__ void isSame(float *array1, float *array2, int width, int height, bool transform)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int size = width * height;
	int r = threadId / width;
	int c = threadId % width;

	//if(threadId > 1024) // for debugging
	//{
	//	return;
	//}

	if(threadId < size)
	{
		if(transform)
		{
			if(fabsf(array1[r * width + c] - array2[c * height + r]) > 1e-3)
			{
				printf("not same at %d, %f ~ %f\n", 
						threadId, array1[r * width + c], array2[c * height + r]);
			}
		}
		else
		{
			if(fabsf(array1[r * width + c] - array2[r * width + c]) > 1e-3)
			{
				printf("not same at %d, %f ~ %f\n", 
						threadId, array1[r * width + c], array2[r * width + c]);
			}
		}
	}
}

int main(int argc, char **argv)
{
    char *image_file = argv[1];
    char *weights_file = argv[2];
    char *bias_file = argv[3];
    char *output_file = argv[4];

	int blockDim = 256;
	int gridDim;

	CNNFunction *func = new CNNCudaFunction();
	func->init();
	func->readImage(image_file);
	func->readParameters(weights_file, bias_file);

	//CNNFunction *funcCudnn = new CNNCudnnFunction();
	//funcCudnn->init();
	//funcCudnn->readImage(image_file);
	//funcCudnn->readParameters(weights_file, bias_file);

    func->convolution(224, 3, 64, 0);
    //funcCudnn->convolution(224, 3, 64, 0);
	
    //func->convolution(4, 2, 1, 0);
    //funcCudnn->convolution(4, 2, 1, 0);

	//checkCudaErrors(hipDeviceSynchronize());
	//gridDim = (4 * 4 * 1 + blockDim - 1) / blockDim;
	//isSame<<<gridDim, blockDim>>>(func->featureOut, funcCudnn->featureOut, 4 * 4, 1, false);
	//checkCudaErrors(hipDeviceSynchronize());
	//return 0;

    func->convolution(224, 64, 64, 1);
    //funcCudnn->convolution(224, 64, 64, 1);

    func->maxpool(224, 64);
    //funcCudnn->maxpool(224, 64);

    func->convolution(112, 64, 128, 2);
    //funcCudnn->convolution(112, 64, 128, 2);

    func->convolution(112, 128, 128, 3);
    //funcCudnn->convolution(112, 128, 128, 3);

    func->maxpool(112, 128);
    //funcCudnn->maxpool(112, 128);

    func->convolution(56, 128, 256, 4);
    //funcCudnn->convolution(56, 128, 256, 4);

    func->convolution(56, 256, 256, 5);
    //funcCudnn->convolution(56, 256, 256, 5);

    func->convolution(56, 256, 256, 6);
    //funcCudnn->convolution(56, 256, 256, 6);

    func->convolution(56, 256, 256, 7);
    //funcCudnn->convolution(56, 256, 256, 7);

    func->maxpool(56, 256);
    //funcCudnn->maxpool(56, 256);

    func->convolution(28, 256, 512, 8);
    //funcCudnn->convolution(28, 256, 512, 8);

    func->convolution(28, 512, 512, 9);
    //funcCudnn->convolution(28, 512, 512, 9);

    func->convolution(28, 512, 512, 10);
    //funcCudnn->convolution(28, 512, 512, 10);
	
    func->convolution(28, 512, 512, 11);
    //funcCudnn->convolution(28, 512, 512, 11);
	
    func->maxpool(28, 512);
    //funcCudnn->maxpool(28, 512);

    func->convolution(14, 512, 512, 12);
    //funcCudnn->convolution(14, 512, 512, 12);

    func->convolution(14, 512, 512, 13);
    //funcCudnn->convolution(14, 512, 512, 13);

    func->convolution(14, 512, 512, 14);
    //funcCudnn->convolution(14, 512, 512, 14);

    func->convolution(14, 512, 512, 15 );
    //funcCudnn->convolution(14, 512, 512, 15 );

    func->maxpool(14, 512);
    //funcCudnn->maxpool(14, 512);

    func->fullyConnected(7, 512, 4096, 16); // most time consuming file input
    //funcCudnn->fullyConnected(7, 512, 4096, 16); // most time consuming file input

    func->fullyConnected(1, 4096, 4096, 17);
    //funcCudnn->fullyConnected(1, 4096, 4096, 17);

    func->fullyConnected(1, 4096, 1000, 18);
    //funcCudnn->fullyConnected(1, 4096, 1000, 18);

    // write 1000 dimension
    func->writeOutput(output_file);

    return 0;
}


