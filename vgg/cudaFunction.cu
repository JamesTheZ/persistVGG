#include "hip/hip_runtime.h"
#include "cudaFunction.h"
#include <cassert>

//void printDArray(float *dArray, int size)
//{
//	float *hArray = new float[size];
//
//	checkCudaErrors(hipMemcpy(hArray, dArray, size * sizeof(float), hipMemcpyDefault));
//
//	for(int i=0; i<size; i++)
//	{
//		printf("%f\n", hArray[i]);
//	}
//	fflush(NULL);
//}

__global__ void convBias(float* fIn, float* filter, 
		const int nFilters, const int nChannels, const int width, 
		float* bias, float* fOut)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= nFilters * width * width)
	{
		return;
	}

	int filterId = tid / (width * width);
	int row = (tid / width) % width;
	int col = tid % width;

	float sum = bias[filterId];
	for(int ch = 0; ch < nChannels; ch++)
	{
		// filter dim is 3
		int i = 0;
		for(int r = row-1; r <= row+1; r++, i++) 
		{
			if(r < 0 || r >= width)
			{
				continue;
			}
			int j = 0;
			for(int c = col-1; c <= col+1; c++, j++)
			{
				if(c < 0 || c >= width)
				{
					continue;
				}
				sum += filter[filterId * nChannels * 9 + ch * 9 + i * 3 + j]
					* fIn[ch * width * width + r * width + c];
			}
		}
	}
	fOut[filterId * width * width + row * width + col] = sum;
}

__global__ void reluForward(float* fIn, float* fOut, const int size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= size)
	{
		return;
	}

	fOut[tid] = fIn[tid] < 0 ? 0 : fIn[tid];
}

// stride: {2, 2}, filterSize: {2, 2}
// make sure that (width % 2 == 0)
__global__ void maxPooling(float *fIn, float *fOut, const int width, const int nChannels)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid >= nChannels * width * width / 4)
	{
		return;
	}

	int outWidth = width / 2;
	int channelId = tid / (outWidth * outWidth);
	int row = (tid / outWidth) % outWidth;
	int col = tid % outWidth;
	int oldRow = row * 2;
	int oldCol = col * 2;

	fOut[channelId * outWidth * outWidth + row * outWidth + col] = fmaxf(
			fmaxf(fIn[channelId * width * width + oldRow * width + oldCol],
				fIn[channelId * width * width + (oldRow+1) * width + oldCol]),
			fmaxf(fIn[channelId * width * width + oldRow * width + (oldCol+1)],
				fIn[channelId * width * width + (oldRow+1) * width + (oldCol+1)]));
}

	template <int BLOCK_HEIGHT, int BLOCK_WIDTH>
__global__ void fullyConnectCUDA(float *fIn, float *filter, 
		int batchSize, int nChannels, int height, int width, int nFilters,
		float *bias, float *fOut)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int wFeature = nChannels * height * width; // width of feature matrix

	if(bx * BLOCK_WIDTH + tx >= nFilters || by * BLOCK_HEIGHT + ty >= batchSize)
	{
		return;
	}

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wFeature * BLOCK_HEIGHT * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd   = aBegin + wFeature - 1;
	aEnd = aEnd <= batchSize - 1 ? aEnd : batchSize - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep  = BLOCK_WIDTH;

	// Index of the first sub-matrix of B processed by the block
	// Note that matrix B should be transformed implicit
	int bBegin = wFeature * BLOCK_HEIGHT * bx;

	// Step size used to iterate through the sub-matrices of B
	// Note that matrix B should be transformed implicit
	int bStep  = BLOCK_WIDTH;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin;
			a <= aEnd;
			a += aStep, b += bStep) {
		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ float As[BLOCK_HEIGHT][BLOCK_WIDTH];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ float Bs[BLOCK_HEIGHT][BLOCK_WIDTH];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = fIn[a + wFeature * ty + tx];
		Bs[ty][tx] = filter[b + wFeature * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_WIDTH; ++k) {
			Csub += As[ty][k] * Bs[tx][k];
			//Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = nFilters * BLOCK_HEIGHT * by + BLOCK_WIDTH * bx;
	fOut[c + nFilters * ty + tx] = Csub + bias[BLOCK_HEIGHT * by + ty];
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
//dim3 threads(block_size, block_size);
//dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);
//    MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
//                                            dimsA.x, dimsB.x);
template <int BLOCK_SIZE> __global__ void matrixMul(
		float *C, float *A, float *B, int wA, int wB) {
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd   = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep  = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep  = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin;
			a <= aEnd;
			a += aStep, b += bStep) {
		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}

void CNNCudaFunction::init()
{
	CNNFunction::init();

	checkCudaErrors(hipdnnCreate(&cudnnHandle));
	checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnIDesc));
	checkCudaErrors(hipdnnCreateFilterDescriptor(&cudnnFDesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnODesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnBiasDesc));

	// all activations in VGGNET are the same.
	checkCudaErrors(hipdnnCreateActivationDescriptor(&cudnnActDesc));
	checkCudaErrors(hipdnnSetActivationDescriptor(cudnnActDesc,
				HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
}

void CNNCudaFunction::fullyConnected(int width, int nChannels, int nFilters, int layerId)
{
	int filterSize = width * width * nChannels;
	float *featureIn = nullptr;
	checkCudaErrors(hipMalloc(&featureIn, filterSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, filterSize * sizeof(float), hipMemcpyDefault));

	const int batchSize = 1; // should be 2^n and lt 256 
	const int blockDimX = 256 / batchSize;
	const int blockDimY = batchSize;
	dim3 threads(blockDimX, blockDimY);
	const int gridDimX = (nFilters + blockDimX - 1) / blockDimX;
	const int gridDimY = (batchSize + blockDimY - 1) / blockDimY;
	dim3 grid(gridDimX, gridDimY);
	fullyConnectCUDA<blockDimY, blockDimX> <<<grid, threads>>>(
			featureIn, weights[layerId], 
			batchSize, nChannels, width, width, nFilters,
			bias[layerId], featureOut);

	//int filterSize = width * width * nChannels;
	//float *featureIn = nullptr;
	//checkCudaErrors(hipMalloc(&featureIn, filterSize * sizeof(float)));
	//checkCudaErrors(hipMemcpy(featureIn, featureOut, filterSize * sizeof(float), hipMemcpyDefault));

	//// CUBLAS is column major, which needs extra transform
	//checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
	//			nFilters, 1, filterSize, &alpha, weights[layerId], filterSize, 
	//			featureIn, filterSize, &beta, featureOut, filterSize));

	//// add bias
	//checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
	//			HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, nFilters, 1, 1));
	//checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnBiasDesc,
	//			HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, nFilters, 1, 1));
	//checkCudaErrors(hipdnnAddTensor(cudnnHandle, 
	//			&alpha, cudnnBiasDesc, bias[layerId], 
	//			&alpha, cudnnODesc, featureOut));

	// activation
	//checkCudaErrors(hipdnnActivationForward(cudnnHandle, cudnnActDesc, 
	//			&alpha, cudnnODesc, featureOut, &beta, cudnnODesc, featureOut));

	// activation: relu
	reluForward<<<(nFilters + 255) / 256, 256>>>(
			featureOut, featureOut, nFilters);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(featureIn));
}

void CNNCudaFunction::maxpool(int width, int nChannels)
{
	float* featureIn = nullptr;
	int featureSize = width * width * nChannels;
	checkCudaErrors(hipMalloc(&featureIn, featureSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, featureSize * sizeof(float), hipMemcpyDefault));

	// only deal with width of even number
	assert(width % 2 == 0);
	int blockDim = 256;
	int gridDim = (nChannels * width * width / 4 + blockDim - 1) / blockDim;
	maxPooling<<<gridDim, blockDim>>>(featureIn, featureOut, width, nChannels);

	checkCudaErrors(hipFree(featureIn));
}

void CNNCudaFunction::convolution(int width, int nChannels, int nFilters, int layerId)
{
	std::size_t inputSize = width * width * nChannels * sizeof(float);
	float* dInput= nullptr;
	checkCudaErrors(hipMalloc(&dInput, inputSize));
	checkCudaErrors(hipMemcpy(dInput, featureOut, inputSize, hipMemcpyDefault));
	float *dFilter = weights[layerId];

	int blockDim = 256;
	int gridDim = (nFilters * width * width + blockDim - 1) / blockDim;
	convBias<<<gridDim, blockDim>>>(dInput, dFilter, nFilters, nChannels, 
			width, bias[layerId], featureOut);

	// activation: relu
	reluForward<<<gridDim, blockDim>>>(featureOut, featureOut, nFilters * width * width);

	checkCudaErrors(hipFree(dInput));
}

