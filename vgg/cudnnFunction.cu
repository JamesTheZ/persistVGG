#include "hip/hip_runtime.h"
#include "cudnnFunction.h"

void CNNCudnnFunction::init()
{
	CNNFunction::init();

    checkCudaErrors(hipdnnCreate(&cudnnHandle));
    checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnIDesc));
	checkCudaErrors(hipdnnCreateFilterDescriptor(&cudnnFDesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnODesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnBiasDesc));

	checkCudaErrors(hipdnnCreateConvolutionDescriptor(&cudnnConvDesc));
	
	// all activations in VGGNET are the same.
	checkCudaErrors(hipdnnCreateActivationDescriptor(&cudnnActDesc));
	checkCudaErrors(hipdnnSetActivationDescriptor(cudnnActDesc,
				HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

	// all poolings in VGGNET are the same.
	checkCudaErrors(hipdnnCreatePoolingDescriptor(&cudnnPoolDesc));
	const int poolDim = 2;
	int windowDim[poolDim] = {2, 2}; 
	int padding[poolDim] = {0, 0}; 
	int stride[poolDim] = {2, 2}; 
	checkCudaErrors(hipdnnSetPoolingNdDescriptor(cudnnPoolDesc,
				HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
				poolDim, windowDim, padding, stride));

	checkCudaErrors(hipMemcpy(featureOut, image, 224 * 224 * 3 * sizeof(float), hipMemcpyDefault));
}

void CNNCudnnFunction::fullyConnected(int width, int channels, int numFilters, int layerId)
{
	int filterSize = width * width * channels;
	float *featureIn = nullptr;
	checkCudaErrors(hipMalloc(&featureIn, filterSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, filterSize * sizeof(float), hipMemcpyDefault));

	// output = filter * featureMap
	// CUBLAS is column major, which needs extra transform
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
				numFilters, 1, filterSize, &alpha, weights[layerId], filterSize, 
				featureIn, filterSize, &beta, featureOut, filterSize));

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(featureIn));

	//int num_weights = (width * width * channels + 1) * num_filters;
	//int filter_size = width * width * channels;
	//float *d_weights = parameters[layerId];

	//float *d_input;
    //size_t input_size = (width * width * channels + 1) * sizeof(float);
    //checkCudaErrors(hipMalloc(&d_input, input_size));

	//if(width == 1)
	//{
	//	checkCudaErrors(hipMemcpy(d_input, featureOut, channels*sizeof(float), hipMemcpyDefault));
	//	float val = 1.0f;
	//	checkCudaErrors(hipMemcpy(d_input + channels, &val, sizeof(float), hipMemcpyDefault));
	//}
	//else
	//{
	//	transformFCCudnn<<< 1, channels >>>(d_input, featureOut, width, channels);
	//}

	//checkCudaErrors(cudnnSgemm(cubHandle, CUDNN_OP_N, 
	//			CUDNN_OP_N, 1, num_filters, filter_size+1,
	//			&alpha, d_input, 1, d_weights, filter_size+1,
	//			&beta, featureOut, 1));

	//checkCudaErrors(hipFree(d_input));
}

void CNNCudnnFunction::maxpool(int width, int channels)
{
	hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
	hipdnnDataType_t type = HIPDNN_DATA_FLOAT;
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnIDesc,
				format, type, 1, channels, width, width));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				format, type, 1, channels, width / 2, width / 2));

	float* featureIn = nullptr;
	int featureSize = width * width * channels;
	checkCudaErrors(hipMalloc(&featureIn, featureSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, featureSize, hipMemcpyDefault));

	checkCudaErrors(hipdnnPoolingForward(cudnnHandle, cudnnPoolDesc,
				&alpha, cudnnIDesc, featureIn, &beta, cudnnODesc, featureOut));

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(featureIn));
	
    //float *d_temp;
    //size_t mem_size = width * width * channels * sizeof(float);
    //checkCudaErrors(hipMalloc(&d_temp, mem_size));
    //checkCudaErrors(hipMemcpy(d_temp, featureOut, mem_size, hipMemcpyDefault));
    //maxpoolingCudnn <<< width / 2, width / 2 >>> (featureOut, d_temp, width, channels);
    //hipFree(d_temp);
}

void CNNCudnnFunction::convolution(int width, int channels, int num_filters, int layerId)
{
	hipdnnDataType_t type = HIPDNN_DATA_FLOAT;
	hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnIDesc,
				format, type, 1, channels, width, width));
	checkCudaErrors(hipdnnSetFilter4dDescriptor(cudnnFDesc,
				type, format, num_filters, channels, 3, 3));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				format, type, 1, num_filters, width, width));

	checkCudaErrors(hipdnnSetConvolution2dDescriptor(cudnnConvDesc,
				1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnConvolutionFwdAlgo_t cudnnConvFwdAlgo;
	checkCudaErrors(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
				cudnnIDesc, cudnnFDesc, cudnnConvDesc, cudnnODesc, 
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &cudnnConvFwdAlgo));

	std::size_t workspaceSize = 0;
	checkCudaErrors(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
				cudnnIDesc, cudnnFDesc, cudnnConvDesc, cudnnODesc,
				cudnnConvFwdAlgo, &workspaceSize));

	float* dWorkspace = nullptr;
	checkCudaErrors(hipMalloc(&dWorkspace, workspaceSize));

	std::size_t inputSize = width * width * channels * sizeof(float);
	float* dInput= nullptr;
    checkCudaErrors(hipMalloc(&dInput, inputSize));
	// memcpy and activation: relu
	checkCudaErrors(hipdnnActivationForward(cudnnHandle, cudnnActDesc, 
				&alpha, cudnnIDesc, featureOut, &beta, cudnnIDesc, dInput));

    float *dFilter = weights[layerId];

	checkCudaErrors(hipdnnConvolutionForward(cudnnHandle,
				&alpha, cudnnIDesc, dInput, cudnnFDesc, dFilter, 
				cudnnConvDesc, cudnnConvFwdAlgo, dWorkspace, workspaceSize,
				&beta, cudnnODesc, featureOut));
	
	// add bias
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnBiasDesc,
				format, type, 1, num_filters, 1, 1));
	checkCudaErrors(hipdnnAddTensor(cudnnHandle, 
				&alpha, cudnnBiasDesc, bias[layerId], 
				&alpha, cudnnODesc, featureOut));

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(dInput));
	checkCudaErrors(hipFree(dWorkspace));

	//checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnIDesc));
	//checkCudaErrors(hipdnnCreateFilterDescriptor(&cudnnFDesc));
	//checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnODesc));

    //int num_weights = (3 * 3 * channels + 1) * num_filters;
    //int output_size = width * width * num_filters;
    //int filter_size = 3 * 3 * channels;
    //int hidden_width = 3 * 3 * channels + 1;

    //float *d_raw_input;
    //float *d_input;
    //size_t input_size = width * width * hidden_width * sizeof(float);
    //checkCudaErrors(hipMalloc(&d_input, input_size));
    //checkCudaErrors(hipMemset(d_input, 0, input_size));
    //// expand original input to (width * width) * (3 * 3 * channels + 1) with a 1 at last for bias
    //if (channels == 3) 
	//{
	//	size_t raw_input_size = width * width * channels * sizeof(float);
    //    checkCudaErrors(hipMemcpy(featureOut, image, raw_input_size, hipMemcpyHostToDevice));
    //    transformImageCudnn <<< width, width >>> (d_input, featureOut, width, channels);
	//}
	//else
	//{
	//	// d_output has width*width rows and channels cols.
    //    transformCudnn <<< width, width >>> (d_input, featureOut, width, channels);
	//}

    //float *d_weights = parameters[layerId];
    //// input * weights = ((width * width) * (3 * 3 * channels + 1)) * ((3 * 3 * channels + 1) * num_filters)
    //checkCudaErrors(cudnnSgemm(cubHandle, CUDNN_OP_N, HIPBLAS_OP_N, 
	//			num_filters, width * width, hidden_width,
    //            &alpha, d_weights, num_filters, d_input, hidden_width,
    //            &beta, featureOut, num_filters));
	//// d_output has width*width rows and num_filters cols.
	//
	//checkCudaErrors(hipFree(d_input));
}

/*
__global__ void transformImageCudnn(float *input, const float *raw_input, const int width, const int channels)
{
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int start_i = thread_id / width - 1;
	int start_j = thread_id % width - 1;
	int per_channel_width = width * width;
	int hidden_width = 3 * 3 * channels + 1;
	int global_offset = thread_id * hidden_width;

	for (int c = 0; c < channels; c++) {
		int offset = 0;
		for (int i = start_i; i < start_i + 3; i++) {
			if (i < 0 || i == width)
				continue;
			for (int j = start_j; j < start_j + 3; j++) {
				if (j < 0 || j == width)
					continue;
				input[global_offset + c * 9 + offset] 
					= raw_input[c * per_channel_width + i * width + j];
				offset++;
			}
		}
		// padding ?? added by Zhen
		// while(offset < 9)
		// {
		// 	input[offset++] = 0;
		// }
	}
	input[(thread_id + 1) * hidden_width - 1] = 1;
}

__global__ void transformFCCudnn(float *input, const float *raw_input, const int width, const int channels)
{
	int thread_id = threadIdx.x;
	int size = width * width;

	for (int s = 0; s < size; s++)
	{
		input[thread_id * size + s] 
			= raw_input[s * channels + thread_id];
	}
	if (thread_id == 0)
	{
		input[width * width * channels] = 1;
	}
}

__global__ void transformCudnn(float *input, const float *raw_input, const int width, const int channels)
{
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int start_i = thread_id / width - 1;
	int start_j = thread_id % width - 1;
	int hidden_width = 3 * 3 * channels + 1;
	int global_offset = thread_id * hidden_width;

	float relu;
	for (int c = 0; c < channels; c++) {
		int offset = 0;
		for (int i = start_i; i < start_i + 3; i++) {
			if (i < 0 || i == width)
				continue;
			for (int j = start_j; j < start_j + 3; j++) {
				if (j < 0 || j == width)
					continue;
				relu = raw_input[(i * width + j) * channels + c];
				input[global_offset + c * 9 + offset] = relu < 0 ? 0 : relu;
				offset++;
			}
		}
		// padding, is this correct ?? added by Zhen
		// while(offset < 9)
		// {
		// 	input[offset++] = 0;
		// }
	}
	input[(thread_id + 1) * hidden_width - 1] = 1;
}

__global__ void maxpoolingCudnn(float *output, const float *input, const int width, const int channels)
{
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int new_width = width / 2;
	int i = thread_id / new_width * 2;
	int j = thread_id % new_width * 2;
	int index = i * width + j;

	for (int c = 0; c < channels; c++) {
		float max = 0;
		if (max < input[index * channels + c])
			max = input[index * channels + c];
		if (max < input[(index + 1) * channels + c])
			max = input[(index + 1) * channels + c];
		if (max < input[(index + width) * channels + c])
			max = input[(index + width) * channels + c];
		if (max < input[(index + width + 1) * channels + c])
			max = input[(index + width + 1) * channels + c];
		output[thread_id * channels + c] = max;
	}
}
*/

