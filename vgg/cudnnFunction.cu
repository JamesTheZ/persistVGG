#include "cudnnFunction.h"

void printDArray(float *dArray, int size)
{
	float *hArray = new float[size];

	checkCudaErrors(hipMemcpy(hArray, dArray, size * sizeof(float), hipMemcpyDefault));

	for(int i=0; i<size; i++)
	{
		printf("%f\n", hArray[i]);
	}
	fflush(NULL);
}

void CNNCudnnFunction::init()
{
	CNNFunction::init();

    checkCudaErrors(hipdnnCreate(&cudnnHandle));
    checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnIDesc));
	checkCudaErrors(hipdnnCreateFilterDescriptor(&cudnnFDesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnODesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnBiasDesc));

	checkCudaErrors(hipdnnCreateConvolutionDescriptor(&cudnnConvDesc));
	
	// all activations in VGGNET are the same.
	checkCudaErrors(hipdnnCreateActivationDescriptor(&cudnnActDesc));
	checkCudaErrors(hipdnnSetActivationDescriptor(cudnnActDesc,
				HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

	// all poolings in VGGNET are the same.
	checkCudaErrors(hipdnnCreatePoolingDescriptor(&cudnnPoolDesc));
	const int poolDim = 2;
	int windowDim[poolDim] = {2, 2}; 
	int padding[poolDim] = {0, 0}; 
	int stride[poolDim] = {2, 2}; 
	checkCudaErrors(hipdnnSetPoolingNdDescriptor(cudnnPoolDesc,
				HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
				poolDim, windowDim, padding, stride));
}

void CNNCudnnFunction::fullyConnected(int width, int nChannels, int nFilters, int layerId)
{
	int filterSize = width * width * nChannels;
	float *featureIn = nullptr;
	checkCudaErrors(hipMalloc(&featureIn, filterSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, filterSize * sizeof(float), hipMemcpyDefault));

	// CUBLAS is column major, which needs extra transform
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
				nFilters, 1, filterSize, &alpha, weights[layerId], filterSize, 
				featureIn, filterSize, &beta, featureOut, filterSize));

	// add bias
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, nFilters, 1, 1));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnBiasDesc,
				HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, nFilters, 1, 1));
	checkCudaErrors(hipdnnAddTensor(cudnnHandle, 
				&alpha, cudnnBiasDesc, bias[layerId], 
				&alpha, cudnnODesc, featureOut));

	// activation
	checkCudaErrors(hipdnnActivationForward(cudnnHandle, cudnnActDesc, 
				&alpha, cudnnODesc, featureOut, &beta, cudnnODesc, featureOut));

	checkCudaErrors(hipFree(featureIn));
}

void CNNCudnnFunction::maxpool(int width, int nChannels)
{
	hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
	hipdnnDataType_t type = HIPDNN_DATA_FLOAT;
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnIDesc,
				format, type, 1, nChannels, width, width));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				format, type, 1, nChannels, width / 2, width / 2));

	float* featureIn = nullptr;
	int featureSize = width * width * nChannels;
	checkCudaErrors(hipMalloc(&featureIn, featureSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, featureSize * sizeof(float), hipMemcpyDefault));

	checkCudaErrors(hipdnnPoolingForward(cudnnHandle, cudnnPoolDesc,
				&alpha, cudnnIDesc, featureIn, &beta, cudnnODesc, featureOut));

	checkCudaErrors(hipFree(featureIn));
}

void CNNCudnnFunction::convolution(int width, int nChannels, int nFilters, int layerId)
{
	hipdnnDataType_t type = HIPDNN_DATA_FLOAT;
	hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnIDesc,
				format, type, 1, nChannels, width, width));
	checkCudaErrors(hipdnnSetFilter4dDescriptor(cudnnFDesc,
				type, format, nFilters, nChannels, 3, 3));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				format, type, 1, nFilters, width, width));

	checkCudaErrors(hipdnnSetConvolution2dDescriptor(cudnnConvDesc,
				1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnConvolutionFwdAlgo_t cudnnConvFwdAlgo;
	checkCudaErrors(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
				cudnnIDesc, cudnnFDesc, cudnnConvDesc, cudnnODesc, 
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &cudnnConvFwdAlgo));

	std::size_t workspaceSize = 0;
	checkCudaErrors(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
				cudnnIDesc, cudnnFDesc, cudnnConvDesc, cudnnODesc,
				cudnnConvFwdAlgo, &workspaceSize));

	float* dWorkspace = nullptr;
	checkCudaErrors(hipMalloc(&dWorkspace, workspaceSize));

	std::size_t inputSize = width * width * nChannels * sizeof(float);
	float* dInput= nullptr;
    checkCudaErrors(hipMalloc(&dInput, inputSize));
	checkCudaErrors(hipMemcpy(dInput, featureOut, inputSize, hipMemcpyDefault));

    float *dFilter = weights[layerId];

	checkCudaErrors(hipdnnConvolutionForward(cudnnHandle,
				&alpha, cudnnIDesc, dInput, cudnnFDesc, dFilter, 
				cudnnConvDesc, cudnnConvFwdAlgo, dWorkspace, workspaceSize,
				&beta, cudnnODesc, featureOut));
	
	// add bias
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnBiasDesc,
				format, type, 1, nFilters, 1, 1));
	checkCudaErrors(hipdnnAddTensor(cudnnHandle, 
				&alpha, cudnnBiasDesc, bias[layerId], 
				&alpha, cudnnODesc, featureOut));

	// activation
	checkCudaErrors(hipdnnActivationForward(cudnnHandle, cudnnActDesc, 
				&alpha, cudnnODesc, featureOut, &beta, cudnnODesc, featureOut));

	checkCudaErrors(hipFree(dInput));
	checkCudaErrors(hipFree(dWorkspace));
}

