#include "cudnnFunction.h"

void printDArray(float *dArray, int size)
{
	float *hArray = new float[size];

	checkCudaErrors(hipMemcpy(hArray, dArray, size * sizeof(float), hipMemcpyDefault));

	for(int i=0; i<size; i++)
	{
		printf("%f\n", hArray[i]);
	}


	fflush(NULL);
}

void CNNCudnnFunction::init()
{
	CNNFunction::init();

    checkCudaErrors(hipdnnCreate(&cudnnHandle));
    checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnIDesc));
	checkCudaErrors(hipdnnCreateFilterDescriptor(&cudnnFDesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnODesc));
	checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnBiasDesc));

	checkCudaErrors(hipdnnCreateConvolutionDescriptor(&cudnnConvDesc));
	
	// all activations in VGGNET are the same.
	checkCudaErrors(hipdnnCreateActivationDescriptor(&cudnnActDesc));
	checkCudaErrors(hipdnnSetActivationDescriptor(cudnnActDesc,
				HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

	// all poolings in VGGNET are the same.
	checkCudaErrors(hipdnnCreatePoolingDescriptor(&cudnnPoolDesc));
	const int poolDim = 2;
	int windowDim[poolDim] = {2, 2}; 
	int padding[poolDim] = {0, 0}; 
	int stride[poolDim] = {2, 2}; 
	checkCudaErrors(hipdnnSetPoolingNdDescriptor(cudnnPoolDesc,
				HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
				poolDim, windowDim, padding, stride));
}

void CNNCudnnFunction::fullyConnected(int width, int numChannels, int numFilters, int layerId)
{
	int filterSize = width * width * numChannels;
	float *featureIn = nullptr;
	checkCudaErrors(hipMalloc(&featureIn, filterSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, filterSize * sizeof(float), hipMemcpyDefault));

	// output = filter * featureMap
	// CUBLAS is column major, which needs extra transform
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
				numFilters, 1, filterSize, &alpha, weights[layerId], filterSize, 
				featureIn, filterSize, &beta, featureOut, filterSize));

	// add bias
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, numFilters, 1, 1));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnBiasDesc,
				HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, numFilters, 1, 1));
	checkCudaErrors(hipdnnAddTensor(cudnnHandle, 
				&alpha, cudnnBiasDesc, bias[layerId], 
				&alpha, cudnnODesc, featureOut));

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(featureIn));

	//int num_weights = (width * width * numChannels + 1) * numFilters;
	//int filter_size = width * width * numChannels;
	//float *d_weights = parameters[layerId];

	//float *d_input;
    //size_t input_size = (width * width * numChannels + 1) * sizeof(float);
    //checkCudaErrors(hipMalloc(&d_input, input_size));

	//if(width == 1)
	//{
	//	checkCudaErrors(hipMemcpy(d_input, featureOut, numChannels*sizeof(float), hipMemcpyDefault));
	//	float val = 1.0f;
	//	checkCudaErrors(hipMemcpy(d_input + numChannels, &val, sizeof(float), hipMemcpyDefault));
	//}
	//else
	//{
	//	transformFCCudnn<<< 1, numChannels >>>(d_input, featureOut, width, numChannels);
	//}

	//checkCudaErrors(cudnnSgemm(cubHandle, CUDNN_OP_N, 
	//			CUDNN_OP_N, 1, numFilters, filter_size+1,
	//			&alpha, d_input, 1, d_weights, filter_size+1,
	//			&beta, featureOut, 1));

	//checkCudaErrors(hipFree(d_input));
}

void CNNCudnnFunction::maxpool(int width, int numChannels)
{
	hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
	hipdnnDataType_t type = HIPDNN_DATA_FLOAT;
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnIDesc,
				format, type, 1, numChannels, width, width));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				format, type, 1, numChannels, width / 2, width / 2));

	float* featureIn = nullptr;
	int featureSize = width * width * numChannels;
	checkCudaErrors(hipMalloc(&featureIn, featureSize * sizeof(float)));
	checkCudaErrors(hipMemcpy(featureIn, featureOut, featureSize, hipMemcpyDefault));

	checkCudaErrors(hipdnnPoolingForward(cudnnHandle, cudnnPoolDesc,
				&alpha, cudnnIDesc, featureIn, &beta, cudnnODesc, featureOut));

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(featureIn));
	
    //float *d_temp;
    //size_t mem_size = width * width * numChannels * sizeof(float);
    //checkCudaErrors(hipMalloc(&d_temp, mem_size));
    //checkCudaErrors(hipMemcpy(d_temp, featureOut, mem_size, hipMemcpyDefault));
    //maxpoolingCudnn <<< width / 2, width / 2 >>> (featureOut, d_temp, width, numChannels);
    //hipFree(d_temp);
}

void CNNCudnnFunction::convolution(int width, int numChannels, int numFilters, int layerId)
{
	hipdnnDataType_t type = HIPDNN_DATA_FLOAT;
	hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnIDesc,
				format, type, 1, numChannels, width, width));
	checkCudaErrors(hipdnnSetFilter4dDescriptor(cudnnFDesc,
				type, format, numFilters, numChannels, 3, 3));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnODesc,
				format, type, 1, numFilters, width, width));

	checkCudaErrors(hipdnnSetConvolution2dDescriptor(cudnnConvDesc,
				1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnConvolutionFwdAlgo_t cudnnConvFwdAlgo;
	checkCudaErrors(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
				cudnnIDesc, cudnnFDesc, cudnnConvDesc, cudnnODesc, 
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &cudnnConvFwdAlgo));

	std::size_t workspaceSize = 0;
	checkCudaErrors(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
				cudnnIDesc, cudnnFDesc, cudnnConvDesc, cudnnODesc,
				cudnnConvFwdAlgo, &workspaceSize));

	float* dWorkspace = nullptr;
	checkCudaErrors(hipMalloc(&dWorkspace, workspaceSize));

	std::size_t inputSize = width * width * numChannels * sizeof(float);
	float* dInput= nullptr;
    checkCudaErrors(hipMalloc(&dInput, inputSize));

	// memcpy and activation: relu
	if(layerId != 0)
	{
		checkCudaErrors(hipdnnActivationForward(cudnnHandle, cudnnActDesc, 
					&alpha, cudnnIDesc, featureOut, &beta, cudnnIDesc, dInput));
	}
	else
	{
		checkCudaErrors(hipMemcpy(dInput, featureOut, inputSize, hipMemcpyDefault));
	}

    float *dFilter = weights[layerId];

	//printDArray(dInput, width * width * numChannels);

	//exit(0);

	checkCudaErrors(hipdnnConvolutionForward(cudnnHandle,
				&alpha, cudnnIDesc, dInput, cudnnFDesc, dFilter, 
				cudnnConvDesc, cudnnConvFwdAlgo, dWorkspace, workspaceSize,
				&beta, cudnnODesc, featureOut));
	
	// add bias
	checkCudaErrors(hipdnnSetTensor4dDescriptor(cudnnBiasDesc,
				format, type, 1, numFilters, 1, 1));
	checkCudaErrors(hipdnnAddTensor(cudnnHandle, 
				&alpha, cudnnBiasDesc, bias[layerId], 
				&alpha, cudnnODesc, featureOut));

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(dInput));
	checkCudaErrors(hipFree(dWorkspace));

	//checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnIDesc));
	//checkCudaErrors(hipdnnCreateFilterDescriptor(&cudnnFDesc));
	//checkCudaErrors(hipdnnCreateTensorDescriptor(&cudnnODesc));

    //int num_weights = (3 * 3 * numChannels + 1) * numFilters;
    //int output_size = width * width * numFilters;
    //int filter_size = 3 * 3 * numChannels;
    //int hidden_width = 3 * 3 * numChannels + 1;

    //float *d_raw_input;
    //float *d_input;
    //size_t input_size = width * width * hidden_width * sizeof(float);
    //checkCudaErrors(hipMalloc(&d_input, input_size));
    //checkCudaErrors(hipMemset(d_input, 0, input_size));
    //// expand original input to (width * width) * (3 * 3 * numChannels + 1) with a 1 at last for bias
    //if (numChannels == 3) 
	//{
	//	size_t raw_input_size = width * width * numChannels * sizeof(float);
    //    checkCudaErrors(hipMemcpy(featureOut, image, raw_input_size, hipMemcpyHostToDevice));
    //    transformImageCudnn <<< width, width >>> (d_input, featureOut, width, numChannels);
	//}
	//else
	//{
	//	// d_output has width*width rows and numChannels cols.
    //    transformCudnn <<< width, width >>> (d_input, featureOut, width, numChannels);
	//}

    //float *d_weights = parameters[layerId];
    //// input * weights = ((width * width) * (3 * 3 * numChannels + 1)) * ((3 * 3 * numChannels + 1) * numFilters)
    //checkCudaErrors(cudnnSgemm(cubHandle, CUDNN_OP_N, HIPBLAS_OP_N, 
	//			numFilters, width * width, hidden_width,
    //            &alpha, d_weights, numFilters, d_input, hidden_width,
    //            &beta, featureOut, numFilters));
	//// d_output has width*width rows and numFilters cols.
	//
	//checkCudaErrors(hipFree(d_input));
}

